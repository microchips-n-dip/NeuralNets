#include "hip/hip_runtime.h"
#if CUDA_SUPPORT 1
__global__ void cudaNeurons(Network** network)
{
    int n = threadIdx.x + (blockIdx.x * blockDimx.x);
    (*(**network).getNeuron(n)).update();
}
	
__global__ void cudaSynapses(Network** network)
{
    int s = threadIdx.x + (blockId.x * blockDimx.x);
    (*(**network).getSynapse(s)).transmit();
}

Network::cudaPropagate()
{
    Network** dev_net;
    int size = sizeof(Network*);
    hipMemcpy(dev_net, &this, size, hipMemcpyHostToDevice);
    cudaSynapses<<<getNeuronCount()/TPB_NET, TBP_NET>>>(dev_net);
    cudaNeurons<<<getNeuronCount()/TPB_NET, TBP_NET>>>(dev_net);
}
#endif
