#include "hip/hip_runtime.h"
#if CUDA_SUPPORT 1
#define TPB_NET 512
__global__ void cudaNeurons(Network** network)
{
    int n = threadIdx.x + (blockIdx.x * blockDim.x);
    (*(**network).getNeuron(n)).update();
}
	
__global__ void cudaSynapses(Network** network)
{
    int s = threadIdx.x + (blockIdx.x * blockDim.x);
    (*(**network).getSynapse(s)).transmit();
}

Network::cudaPropagate()
{
    Network** dev_net;
    int size = sizeof(Network*);
    hipMemcpy(dev_net, &this, size, hipMemcpyHostToDevice);
    cudaSynapses<<<getNeuronCount()/TPB_NET, TPB_NET>>>(dev_net);
    cudaNeurons<<<getNeuronCount()/TPB_NET, TPB_NET>>>(dev_net);
}
#endif
